
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define I(row, col, ncols) (row = ncols + col)

#define CUDA_CALL(x) {if ((x) != hipSuccess) { \
    printf("CUDA error at %s:%d\n", __FILE__, __LINE__); \
    printf("   %s\n", hipGetErrorString(hipGetLastError())); \
    exit (EXIT_FAILURE);\
}}

__global__ void get_dst (float *dst, float *x, float *y,
                         float *mu_x, float *mu_y) {
    int i = blockIdx.x;
    int j = threadIdx.x;

    dst[I(i, j, blockDim.x)] = (x[i] - mu_x[j]) * (x[i] - mu_x[j]);
    dst[I(i, j, blockDim.x)] += (y[i] - mu_y[j]) * (y[i] - mu_y[j]);
}

__global__ void regroup (int *group, float *dst, int k) {
    int i = blockIdx.x;
    int j;
    float min_dst;

    min_dst = dst[I(i, 0, k)];
    group[i] = 1;
    for (j = 1; j < k; ++ j) {
        if (dst[I(i, j, k)] < min_dst) {
            min_dst = dst[I(i, j, k)];
            group[i] = j + 1;
        }
    }
}

__global__ void clear (float *sum_x, float *sum_y, int *nx, int *ny) {
    int j = threadIdx.x;
    sum_x[j] = 0;
    sum_y[j] = 0;
    nx[j] = 0;
    ny[j] = 0;
}

__global__ void recenter_step1(float *sum_x, float *sum_y, int *nx, int *ny,
                               float *x, float *y, int *group, int n) {
    int j = threadIdx.x;

    for (int i = 0; i < n; ++ i) {
        if (group[i] == (j + 1)) {
            sum_x[j] += x[i];
            sum_y[j] += y[i];
            nx[j] ++;
            ny[j] ++;
        }
    }
}

__global__ void recenter_step2(float *mu_x, float *mu_y, float *sum_x,
                               float *sum_y, int *nx, int *ny) {
    int j = threadIdx.x;
    mu_x[j] = sum_x[j] / nx[j];
    mu_y[j] = sum_y[j] / ny[j];
}

void kmeans (int nreps, int n, int k,
             float *x_d, float *y_d, float *mu_x_d, float *mu_y_d,
             int *group_d, int *nx_d, int *ny_d,
             float *sum_x_d, float *sum_y_d, float *dst_d) {
    int i;
    for (i = 0; i < nreps; ++ i) {
        get_dst<<<n, k>>>(dst_d, x_d, y_d, mu_x_d, mu_y_d);
        regroup<<<n, 1>>>(group_d, dst_d, k);
        clear<<<1, k>>>(sum_x_d, sum_y_d, nx_d, ny_d);
        recenter_step1<<<1, k>>>(sum_x_d, sum_x_d, nx_d, ny_d, x_d, y_d,
                group_d, n);
        recenter_step2<<<1, k>>>(mu_x_d, mu_y_d, sum_x_d, sum_y_d, nx_d,
                ny_d);
    }
}

void read_data(float **x, float **y, float **mu_x, float **my_y,
               int *n, int *k);
void print_results(int *group, float *mu_x, float *my_y, int n, int k);

int main () {
    int n; /*样本点的个数*/
    int k; /*最后聚成k类*/
    int *group;
    float *x = nullptr, *y = nullptr, *mu_x = nullptr, *mu_y = nullptr;

    /*gpu variables*/
    int *group_d, *mx_d, *my_d;
    float *x_d, *y_d, *mu_x_d, *mu_y_d, *sum_x_d, *sum_y_d, *dst_d;

    /*read data from files on cpu*/
    read_data(&x, &y, &mu_x, &mu_y, &n, &k);

    /*allocate cpu memory*/
    group = (int *) malloc (n * sizeof (int));

    /* allocate gpu memory*/
    CUDA_CALL(hipMalloc((void**) &group_d, n * sizeof (int)));
    CUDA_CALL(hipMalloc((void**) &mx_d, k * sizeof (int)));
    CUDA_CALL(hipMalloc((void**) &my_d, k * sizeof (int )));
    CUDA_CALL(hipMalloc((void**) &x_d, n * sizeof (int)));
    CUDA_CALL(hipMalloc((void**) &y_d, n * sizeof (int)));
    CUDA_CALL(hipMalloc((void**) &mu_x_d, k * sizeof (float)));
    CUDA_CALL(hipMalloc((void**) &mu_y_d, k * sizeof (float)));
    CUDA_CALL(hipMalloc((void**) &sum_x_d, k * sizeof (float)));
    CUDA_CALL(hipMalloc((void**) &sum_y_d, k * sizeof (float)));
    CUDA_CALL(hipMalloc((void**) &dst_d, n * k * sizeof (float)));

    /* write data to gpu*/
    CUDA_CALL(hipMemcpy(x_d, x, n * sizeof (float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(y_d, y, n * sizeof (float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(mu_x_d, mu_x, k * sizeof (float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(mu_y_d, mu_y, k * sizeof (float), hipMemcpyHostToDevice));

    /*perform kmeans*/
    kmeans(10, n, k, x_d, y_d, mu_x_d, mu_y_d, group_d, mx_d, my_d,
           sum_x_d, sum_y_d, dst_d);

    /*read back data from gpu*/
    CUDA_CALL(hipMemcpy(group, group_d, n * sizeof (int), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(mu_x, mu_x_d, k * sizeof (float), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(mu_y, mu_y_d, k * sizeof (float), hipMemcpyDeviceToHost));

    /*prin results and clean up*/
    print_results(group, mu_x, mu_y, n, k);

    free(x);
    free(y);
    free(mu_x);
    free(mu_y);
    free(group);

    CUDA_CALL(hipFree(x_d));
    CUDA_CALL(hipFree(y_d));
    CUDA_CALL(hipFree(mu_x_d));
    CUDA_CALL(hipFree(mu_y_d));
    CUDA_CALL(hipFree(group_d));
    CUDA_CALL(hipFree(mx_d));
    CUDA_CALL(hipFree(my_d));
    CUDA_CALL(hipFree(sum_x_d));
    CUDA_CALL(hipFree(sum_y_d));
    CUDA_CALL(hipFree(dst_d));

    return 0;
}
